#include "hip/hip_runtime.h"
/*
 * Keccak 256
 *
 */

extern "C"
{
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_keccak.h"

#include "miner.h"
}

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void keccak256_cpu_free(int thr_id);
extern void keccak256_setBlock_80(void *pdata,const void *ptarget);
extern uint32_t keccak256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

// CPU Hash
extern "C" void keccak256_hash(void *state, const void *input)
{
	uint32_t _ALIGN(64) hash[16];
	sph_keccak_context ctx_keccak;

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256 (&ctx_keccak, input, 80);
	sph_keccak256_close(&ctx_keccak, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_keccak256(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << 21); // 256*256*8*4
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], throughput * 64));
		keccak256_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 19; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

	keccak256_setBlock_80((void*)endiandata, ptarget);
	do {
		int order = 0;

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = keccak256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		if (work->nonces[0] != UINT32_MAX && bench_algo < 0)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];

			be32enc(&endiandata[19], work->nonces[0]);
			keccak256_hash(vhash, endiandata);

			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				pdata[19] = work->nonces[0] + 1;
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_keccak256(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	keccak256_cpu_free(thr_id);

	hipDeviceSynchronize();
	init[thr_id] = false;
}
